
#include <hip/hip_runtime.h>
#define TILE_DIM 16
extern "C" __global__ void dot_product(float *v1, float *v2, float *o, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid]=0;
    __syncthreads();    
    if (i < n) {
        sdata[tid] = v1[i] * v2[i];
    }
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        o[blockIdx.x] = sdata[0];
    }
}


extern "C" __global__ void reduce(float *v1, float *o, int n) {
    extern __shared__ float sdata2[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata2[tid]=0;
    __syncthreads();    
   
    if (i < n) {
        sdata2[tid] = v1[i];
    }
    __syncthreads();

    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata2[tid] += sdata2[tid + s];
        }
        __syncthreads();
    }

    
    if (tid == 0) {
        o[blockIdx.x] = sdata2[0];
    }
}
extern "C" __global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, int BRows,
    int BCols, int CRows, int CCols)
{
    float CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

         if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
             As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
             Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n)
             CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < CRows && Col < CCols)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
           (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
}
extern "C" __global__ void transpose(float *in, float *out, unsigned int nx, unsigned int ny){
	unsigned int ix=blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int iy=blockDim.y * blockIdx.y + threadIdx.y;
	if (ix>=nx || iy>=ny) return;
	out[iy*nx + ix]=in[ix*ny + iy];
}